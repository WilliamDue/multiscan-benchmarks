#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "../../common/sps.cu.h"
#include "../../common/util.cu.h"
#include "../../common/data.h"
#include <math.h>

using token_t = uint8_t;
using state_t = uint16_t;

const uint32_t NUM_STATES = 12;
const uint32_t NUM_TRANS = 256;
// const token_t IGNORE_TOKEN = 0;
const state_t ENDO_MASK = 15;
const state_t ENDO_OFFSET = 0;
const state_t TOKEN_MASK = 112;
const state_t TOKEN_OFFSET = 4;
const state_t ACCEPT_MASK = 128;
const state_t ACCEPT_OFFSET = 7;
const state_t PRODUCE_MASK = 256;
const state_t PRODUCE_OFFSET = 8;
const state_t IDENTITY = 74;

state_t h_to_state[NUM_TRANS] =
        {75, 75, 75, 75, 75, 75, 75, 75, 75, 128, 128, 75, 75, 128,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 128, 75, 75, 75, 75, 75, 75, 75, 161, 178, 75,
         75, 75, 75, 75, 75, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 75, 75, 75, 75, 75, 75, 75, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 75, 75,
         75, 75, 75, 75, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75};

state_t h_compose[NUM_STATES * NUM_STATES] =
    {132, 392, 392, 392, 132, 392, 392, 392, 132, 392, 128, 75,
     421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 161, 75,
     438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 178, 75,
     407, 407, 407, 153, 407, 407, 407, 153, 407, 153, 147, 75,
     132, 132, 132, 132, 132, 132, 132, 132, 132, 132, 132, 75,
     421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 75,
     438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 75,
     407, 407, 407, 407, 407, 407, 407, 407, 407, 407, 407, 75,
     392, 392, 392, 392, 392, 392, 392, 392, 392, 392, 392, 75,
     153, 153, 153, 153, 153, 153, 153, 153, 153, 153, 153, 75,
     128, 161, 178, 147, 132, 421, 438, 407, 392, 153, 74, 75,
     75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75};


__device__ __host__ __forceinline__ state_t get_index(state_t state) {
    return (state & ENDO_MASK) >> ENDO_OFFSET;
}

__device__ __host__ __forceinline__ token_t get_token(state_t state) {
    return (state & TOKEN_MASK) >> TOKEN_OFFSET;
}

bool is_accept(state_t state) {
    return (state & ACCEPT_MASK) >> ACCEPT_OFFSET;
}

__device__ __host__ __forceinline__ bool is_produce(state_t state) {
    return (state & PRODUCE_MASK) >> PRODUCE_OFFSET;
}

struct LexerCtx {
    state_t* d_to_state;
    state_t* d_compose;

    LexerCtx() : d_to_state(NULL), d_compose(NULL) {
        hipMalloc(&d_to_state, sizeof(h_to_state));
        hipMemcpy(d_to_state, h_to_state, sizeof(h_to_state),
                hipMemcpyHostToDevice);
        hipMalloc(&d_compose, sizeof(h_compose));
        hipMemcpy(d_compose, h_compose, sizeof(h_compose),
                hipMemcpyHostToDevice);
    }

    void Cleanup() {
        if (d_to_state) hipFree(d_to_state);
        if (d_compose) hipFree(d_compose);
    }

    __device__ __host__ __forceinline__
    state_t operator()(const state_t &a, const state_t &b) const {
        return d_compose[get_index(b) * NUM_STATES + get_index(a)];
    }

    __device__ __host__ __forceinline__
    state_t operator()(const volatile state_t &a, const volatile state_t &b) const {
        return d_compose[get_index(b) * NUM_STATES + get_index(a)];
    }

    __device__ __host__ __forceinline__
    state_t to_state(const char &a) const {
        return d_to_state[a];
    }
};

template<typename I>
struct Add {
    __device__ __forceinline__ I operator()(I a, I b) const {
        return a + b;
    }
};


template<typename I, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
lexer(LexerCtx *ctx,
      uint8_t* d_in,
      uint32_t* d_index_out,
      token_t* d_token_out,
      volatile State<state_t>* state_states,
      volatile State<I>* index_states,
      I size,
      I num_logical_blocks,
      volatile uint32_t* dyn_index_ptr,
      volatile I* new_size) {
    volatile __shared__ state_t states[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ state_t states_aux[BLOCK_SIZE];
    volatile __shared__ I indices[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I indices_aux[BLOCK_SIZE];
    bool is_produce_state[ITEMS_PER_THREAD];


    uint32_t dyn_index = dynamicIndex<uint32_t>(dyn_index_ptr);
    I glb_offs = dyn_index * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            states[lid] = ctx->to_state(d_in[gid]);
        } else {
            states[lid] = IDENTITY;
        }
    }

    __syncthreads();

    scan<state_t, I, LexerCtx, ITEMS_PER_THREAD>(states, states_aux, state_states, *ctx, IDENTITY, dyn_index);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        is_produce_state[i] = gid == I() || is_produce(states[lid]);
        indices[lid] = is_produce_state[i];
    }

    __syncthreads();

    scan<I, I, Add<I>, ITEMS_PER_THREAD>(indices, indices_aux, index_states, Add<I>(), I(), dyn_index);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && is_produce_state[i]) {
            I offset = indices[lid] - 1;
            d_index_out[offset] = gid;
            d_token_out[offset] = get_token(states[lid]);
        }
    }
    
    if (dyn_index == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = indices[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
    }

    __syncthreads();
}

void testLexer(uint8_t* input, size_t input_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I IN_ARRAY_BYTES = size * sizeof(uint8_t);
    const I INDEX_OUT_ARRAY_BYTES = size * sizeof(I);
    const I TOKEN_OUT_ARRAY_BYTES = size * sizeof(token_t);
    const I STATE_STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<state_t>);
    const I INDEX_STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 500;
    const I RUNS = 50;

    std::vector<token_t> h_token_out(size, 0);
    std::vector<I> h_index_out(size, 0);

    uint32_t* d_dyn_index_ptr;
    I* d_new_size;
    uint8_t *d_in;
    I *d_index_out;
    token_t *d_token_out;
    State<I>* d_index_states;
    State<state_t>* d_state_states;
    gpuAssert(hipMalloc((void**)&d_dyn_index_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_index_states, INDEX_STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_state_states, STATE_STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, IN_ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_index_out, INDEX_OUT_ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_token_out, TOKEN_OUT_ARRAY_BYTES));
    gpuAssert(hipMemcpy(d_in, input, IN_ARRAY_BYTES, hipMemcpyHostToDevice));
    
    LexerCtx ctx = LexerCtx();
    
    for (I i = 0; i < WARMUP_RUNS; ++i) {
        lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(&ctx, d_in, d_index_out, d_token_out, d_state_states, d_index_states, size, NUM_LOGICAL_BLOCKS, d_dyn_index_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    timeval * temp = (timeval *) malloc(sizeof(timeval) * RUNS);
    timeval prev;
    timeval curr;
    timeval t_diff;

    for (I i = 0; i < RUNS; ++i) {
        gettimeofday(&prev, NULL);
        lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(&ctx, d_in, d_index_out, d_token_out, d_state_states, d_index_states, size, NUM_LOGICAL_BLOCKS, d_dyn_index_ptr, d_new_size);
        hipDeviceSynchronize();
        gettimeofday(&curr, NULL);
        timeval_subtract(&t_diff, &curr, &prev);
        temp[i] = t_diff;
        hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    const I OUT_WRITE = temp_size * (sizeof(I) + sizeof(token_t));
    const I IN_READ = IN_ARRAY_BYTES;
    const I IN_STATE_MAP = sizeof(state_t) * size;
    const I SCAN_READ =  sizeof(state_t) * (size + size / 2); // Lowerbound, it does more work.
    compute_descriptors(temp, RUNS, IN_READ + IN_STATE_MAP + SCAN_READ + OUT_WRITE);
    free(temp);

    lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(&ctx, d_in, d_index_out, d_token_out, d_state_states, d_index_states, size, NUM_LOGICAL_BLOCKS, d_dyn_index_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_index_out.data(), d_index_out, INDEX_OUT_ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(h_token_out.data(), d_token_out, TOKEN_OUT_ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    /*
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << std::endl;
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Test Failed: Due to elements mismatch at index=" << i << std::endl;
            }
        } 
    }

    if (test_passes) {
        std::cout << "Filter test passed." << std::endl;
    }
    */
    

    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_token_out));
    gpuAssert(hipFree(d_index_out));
    gpuAssert(hipFree(d_index_states));
    gpuAssert(hipFree(d_state_states));
    gpuAssert(hipFree(d_dyn_index_ptr));
    gpuAssert(hipFree(d_new_size));

    ctx.Cleanup();
}

int main(int32_t argc, char *argv[]) {
    assert(argc == 2);
    size_t input_size;
    uint8_t* input = read_file(argv[1], &input_size);
    testLexer(input, input_size);

    free(input);
    gpuAssert(hipPeekAtLastError());
    return 0;
}
