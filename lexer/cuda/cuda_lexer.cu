#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "../../common/sps.cu.h"
#include "../../common/util.cu.h"
#include "../../common/data.h"
#include <math.h>
#define PAD "%-38s "

using token_t = uint8_t;
using state_t = uint16_t;

const uint32_t NUM_STATES = 12;
const uint32_t NUM_TRANS = 256;
// const token_t IGNORE_TOKEN = 0;
const state_t ENDO_MASK = 15;
const state_t ENDO_OFFSET = 0;
const state_t TOKEN_MASK = 112;
const state_t TOKEN_OFFSET = 4;
const state_t ACCEPT_MASK = 128;
const state_t ACCEPT_OFFSET = 7;
const state_t PRODUCE_MASK = 256;
const state_t PRODUCE_OFFSET = 8;
const state_t IDENTITY = 74;

state_t h_to_state[NUM_TRANS] =
        {75, 75, 75, 75, 75, 75, 75, 75, 75, 128, 128, 75, 75, 128,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 128, 75, 75, 75, 75, 75, 75, 75, 161, 178, 75,
         75, 75, 75, 75, 75, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 75, 75, 75, 75, 75, 75, 75, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 75, 75,
         75, 75, 75, 75, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147, 147,
         147, 147, 147, 147, 147, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75,
         75, 75, 75, 75};

state_t h_compose[NUM_STATES * NUM_STATES] =
    {132, 392, 392, 392, 132, 392, 392, 392, 132, 392, 128, 75,
     421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 161, 75,
     438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 178, 75,
     407, 407, 407, 153, 407, 407, 407, 153, 407, 153, 147, 75,
     132, 132, 132, 132, 132, 132, 132, 132, 132, 132, 132, 75,
     421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 421, 75,
     438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 438, 75,
     407, 407, 407, 407, 407, 407, 407, 407, 407, 407, 407, 75,
     392, 392, 392, 392, 392, 392, 392, 392, 392, 392, 392, 75,
     153, 153, 153, 153, 153, 153, 153, 153, 153, 153, 153, 75,
     128, 161, 178, 147, 132, 421, 438, 407, 392, 153, 74, 75,
     75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75, 75};


__device__ __host__ __forceinline__ state_t get_index(state_t state) {
    return (state & ENDO_MASK) >> ENDO_OFFSET;
}

__device__ __host__ __forceinline__ token_t get_token(state_t state) {
    return (state & TOKEN_MASK) >> TOKEN_OFFSET;
}

__device__ bool is_accept(state_t state) {
    return (state & ACCEPT_MASK) >> ACCEPT_OFFSET;
}

__device__ __host__ __forceinline__ bool is_produce(state_t state) {
    return (state & PRODUCE_MASK) >> PRODUCE_OFFSET;
}

struct LexerCtx {
    state_t* d_to_state;
    state_t* d_compose;

    LexerCtx() : d_to_state(NULL), d_compose(NULL) {
        hipMalloc(&d_to_state, sizeof(h_to_state));
        hipMemcpy(d_to_state, h_to_state, sizeof(h_to_state),
                hipMemcpyHostToDevice);
        hipMalloc(&d_compose, sizeof(h_compose));
        hipMemcpy(d_compose, h_compose, sizeof(h_compose),
                hipMemcpyHostToDevice);
    }

    void Cleanup() {
        if (d_to_state) hipFree(d_to_state);
        if (d_compose) hipFree(d_compose);
    }

    __device__ __host__ __forceinline__
    state_t operator()(const state_t &a, const state_t &b) const {
        return d_compose[get_index(b) * NUM_STATES + get_index(a)];
    }

    __device__ __host__ __forceinline__
    state_t operator()(const volatile state_t &a, const volatile state_t &b) const {
        return d_compose[get_index(b) * NUM_STATES + get_index(a)];
    }

    __device__ __host__ __forceinline__
    state_t to_state(const char &a) const {
        return d_to_state[a];
    }
};

template<typename I>
struct Add {
    __device__ __forceinline__ I operator()(I a, I b) const {
        return a + b;
    }
};

template<typename T, typename I, typename OP, I ITEMS_PER_THREAD>
__device__ inline void
decoupledLookbackScanSuffix(volatile State<T>* states,
                            volatile state_t* suffixes,
                            volatile T* shmem,
                            OP op,
                            const T ne,
                            uint32_t dyn_idx) {
    volatile __shared__ T values[WARP];
    volatile __shared__ Status statuses[WARP];
    volatile __shared__ T shmem_prefix;
    const uint8_t lane = threadIdx.x & (WARP - 1);
    const bool is_first = threadIdx.x == 0;

    T aggregate = shmem[ITEMS_PER_THREAD * blockDim.x - 1];

    if (is_first) {
        states[dyn_idx].aggregate = aggregate;
    }
    
    if (dyn_idx == 0 && is_first) {
        states[dyn_idx].prefix = aggregate;
    }
    
    __threadfence();
    if (dyn_idx == 0 && is_first) {
        states[dyn_idx].status = Prefix;
    } else if (is_first) {
        states[dyn_idx].status = Aggregate;
    }

    T prefix = ne;
    if (threadIdx.x < WARP && dyn_idx != 0) {
        I lookback_idx = threadIdx.x + dyn_idx;
        I lookback_warp = WARP;
        Status status = Aggregate;
        do {
            if (lookback_warp <= lookback_idx) {
                I idx = lookback_idx - lookback_warp;
                status = states[idx].status;
                statuses[threadIdx.x] = status;
                values[threadIdx.x] = status == Prefix ? states[idx].prefix : states[idx].aggregate;
            } else {
                statuses[threadIdx.x] = Aggregate;
                values[threadIdx.x] = ne;
            }

            scanWarp<T, I, OP>(values, statuses, op, lane);

            T result = values[WARP - 1];
            status = statuses[WARP - 1];

            if (status == Invalid)
                continue;
                
            if (is_first) {
                prefix = op(result, prefix);
            }

            lookback_warp += WARP;
        } while (status != Prefix);
    }

    if (is_first) {
        shmem_prefix = prefix;
    }

    __syncthreads();

    prefix = shmem_prefix;
    const I offset = threadIdx.x * ITEMS_PER_THREAD;
    const I upper = offset + ITEMS_PER_THREAD;
    #pragma unroll
    for (I lid = offset; lid < upper; lid++) {
        shmem[lid] = op(prefix, shmem[lid]);
    }

    if (is_first) {
        states[dyn_idx].prefix = op(prefix, aggregate);
        suffixes[dyn_idx] = shmem[0]; 
        __threadfence();
        states[dyn_idx].status = Prefix;
    }
    
    __syncthreads();
}

template<typename I, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
lexer(LexerCtx ctx,
      uint8_t* d_in,
      uint32_t* d_index_out,
      token_t* d_token_out,
      volatile state_t* suffixes,
      volatile State<state_t>* state_states,
      volatile State<I>* index_states,
      I size,
      I num_logical_blocks,
      volatile uint32_t* dyn_index_ptr,
      volatile I* new_size,
      volatile bool* is_valid) {
    volatile __shared__ state_t states[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I indices[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I indices_aux[BLOCK_SIZE];
    volatile state_t* states_aux = (volatile state_t*) indices;
    const I REG_MEM = 1 + (ITEMS_PER_THREAD - 1) / sizeof(uint64_t);
    uint64_t copy_reg[REG_MEM];
    uint8_t *chars_reg = (uint8_t*) copy_reg;
    uint32_t is_produce_state = 0;

    uint32_t dyn_index = dynamicIndex<uint32_t>(dyn_index_ptr);
    I glb_offs = dyn_index * BLOCK_SIZE * ITEMS_PER_THREAD;
    
    states_aux[threadIdx.x] = ctx.to_state(threadIdx.x);

    __syncthreads();

    /*
    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            chars_reg[i] = d_in[gid];
        }
    }

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            states[lid] = states_aux[chars_reg[i]];
        } else {
            states[lid] = IDENTITY;
        }
    }
    */

    #pragma unroll
    for (I i = 0; i < REG_MEM; i++) {
        I uint64_lid = i * blockDim.x + threadIdx.x;
        I lid = sizeof(uint64_t) * uint64_lid;
        I gid = glb_offs + lid;
        if (gid + sizeof(uint64_t) < size) {
            copy_reg[i] = *((uint64_t*) (gid + (uint8_t*) d_in));
        } else {
            for (I j = 0; j < sizeof(uint64_t); j++) {
                I loc_gid = gid + j;
                if (loc_gid < size) {
                    chars_reg[sizeof(uint64_t) * i + j] = d_in[loc_gid];
                }
            }
        }
    }
    
    #pragma unroll
    for (I i = 0; i < REG_MEM; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I _gid = glb_offs + sizeof(uint64_t) * lid;
        for (I j = 0; j < sizeof(uint64_t); j++) {
            I gid = _gid + j;
	    I lid_off = sizeof(uint64_t) * lid + j;
	    I reg_off = sizeof(uint64_t) * i + j;
	    bool is_in_block = lid_off < ITEMS_PER_THREAD * BLOCK_SIZE; 
            if (gid < size && is_in_block) {
                states[lid_off] = states_aux[chars_reg[reg_off]];
            } else if (is_in_block) {
                states[lid_off] = IDENTITY;
            }
        }
    }

    __syncthreads();
    
    scanBlock<state_t, I, LexerCtx, ITEMS_PER_THREAD>(states, states_aux, ctx);

    decoupledLookbackScanSuffix<state_t, I, LexerCtx, ITEMS_PER_THREAD>(state_states, suffixes, states, ctx, IDENTITY, dyn_index);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        bool temp = false;
        if (gid < size) {
            if (lid == ITEMS_PER_THREAD * BLOCK_SIZE - 1) {
                while (state_states[dyn_index + 1].status != Prefix)
                    continue;
                
                temp = gid == size - 1 || is_produce(suffixes[dyn_index + 1]);
            } else {
                temp = gid == size - 1 || is_produce(states[lid + 1]);
            }
        }
        is_produce_state |= temp << i;
        indices[lid] = temp;
    }

    __syncthreads();

    scan<I, I, Add<I>, ITEMS_PER_THREAD>(indices, indices_aux, index_states, Add<I>(), I(), dyn_index);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && ((is_produce_state >> i) & 1)) {
            I offset = indices[lid] - 1;
            d_index_out[offset] = gid;
            d_token_out[offset] = get_token(states[lid]);
        }
    }
    
    if (dyn_index == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = indices[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
        *is_valid = is_accept(states[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    }
}

void testLexer(uint8_t* input,
               size_t input_size,
               uint32_t* expected_indices,
               token_t* expected_tokens,
               size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 31;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I IN_ARRAY_BYTES = size * sizeof(uint8_t);
    const I INDEX_OUT_ARRAY_BYTES = size * sizeof(I);
    const I TOKEN_OUT_ARRAY_BYTES = size * sizeof(token_t);
    const I STATE_STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<state_t>);
    const I SUFFIXES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(state_t);
    const I INDEX_STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 500;
    const I RUNS = 50;

    std::vector<token_t> h_token_out(size, 0);
    std::vector<I> h_index_out(size, 0);

    uint32_t* d_dyn_index_ptr;
    I* d_new_size;
    bool* d_is_valid;
    uint8_t *d_in;
    I *d_index_out;
    token_t *d_token_out;
    State<I>* d_index_states;
    State<state_t>* d_state_states;
    state_t* d_suffixes;
    gpuAssert(hipMalloc((void**)&d_dyn_index_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    gpuAssert(hipMalloc((void**)&d_is_valid, sizeof(bool)));
    hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
    hipMemset(d_is_valid, false, sizeof(bool));
    gpuAssert(hipMalloc((void**)&d_index_states, INDEX_STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_suffixes, SUFFIXES_BYTES));
    gpuAssert(hipMalloc((void**)&d_state_states, STATE_STATES_BYTES));
    I padding = IN_ARRAY_BYTES; // sizeof(uint64_t) - (IN_ARRAY_BYTES % sizeof(uint64_t));
    gpuAssert(hipMalloc((void**)&d_in, IN_ARRAY_BYTES + padding));
    gpuAssert(hipMalloc((void**)&d_index_out, INDEX_OUT_ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_token_out, TOKEN_OUT_ARRAY_BYTES));
    gpuAssert(hipMemcpy(d_in, input, IN_ARRAY_BYTES, hipMemcpyHostToDevice));
    
    LexerCtx ctx = LexerCtx();

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            ctx,
            d_in,
            d_index_out,
            d_token_out,
            d_suffixes,
            d_state_states,
            d_index_states,
            size,
            NUM_LOGICAL_BLOCKS,
            d_dyn_index_ptr,
            d_new_size,
            d_is_valid
        );
        hipDeviceSynchronize();
        hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    timeval * temp = (timeval *) malloc(sizeof(timeval) * RUNS);
    timeval prev;
    timeval curr;
    timeval t_diff;

    for (I i = 0; i < RUNS; ++i) {
        gettimeofday(&prev, NULL);
        lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            ctx,
            d_in,
            d_index_out,
            d_token_out,
            d_suffixes,
            d_state_states,
            d_index_states,
            size,
            NUM_LOGICAL_BLOCKS,
            d_dyn_index_ptr,
            d_new_size,
            d_is_valid
        );
        hipDeviceSynchronize();
        gettimeofday(&curr, NULL);
        timeval_subtract(&t_diff, &curr, &prev);
        temp[i] = t_diff;
        hipMemset(d_dyn_index_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    const I OUT_WRITE = temp_size * (sizeof(I) + sizeof(token_t));
    const I IN_READ = IN_ARRAY_BYTES;
    const I IN_STATE_MAP = sizeof(state_t) * size;
    const I SCAN_READ =  sizeof(state_t) * (size + size / 2); // Lowerbound, it does more work.
    
    lexer<I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
        ctx,
        d_in,
        d_index_out,
        d_token_out,
        d_suffixes,
        d_state_states,
        d_index_states,
        size,
        NUM_LOGICAL_BLOCKS,
        d_dyn_index_ptr,
        d_new_size,
        d_is_valid
    );
    hipDeviceSynchronize();
    gpuAssert(hipPeekAtLastError());
    bool is_valid = false;
    gpuAssert(hipMemcpy(h_index_out.data(), d_index_out, INDEX_OUT_ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(h_token_out.data(), d_token_out, TOKEN_OUT_ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&is_valid, d_is_valid, sizeof(bool), hipMemcpyDeviceToHost));
    
    bool test_passes = is_valid;

    if (!test_passes) {
        std::cout << "Lexer Test Failed: The input given to the lexer does not result in an accepting state." << std::endl;
    }

    test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Lexer Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << std::endl;
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_index_out[i] == expected_indices[i];
            test_passes &= h_token_out[i] == expected_tokens[i];

            if (!test_passes) {
                std::cout << "Lexer Test Failed: Due to elements mismatch at index=" << i << std::endl;
                break;
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, IN_READ + IN_STATE_MAP + SCAN_READ + OUT_WRITE);
    }    

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_suffixes));
    gpuAssert(hipFree(d_token_out));
    gpuAssert(hipFree(d_index_out));
    gpuAssert(hipFree(d_index_states));
    gpuAssert(hipFree(d_state_states));
    gpuAssert(hipFree(d_dyn_index_ptr));
    gpuAssert(hipFree(d_new_size));

    ctx.Cleanup();
}

int main(int32_t argc, char *argv[]) {
    assert(argc == 3);
    size_t input_size;
    uint8_t* input = read_u8_file(argv[1], &input_size);
    uint32_t* expected_indices = NULL;
    size_t expected_indices_size = 0;
    uint8_t* expected_tokens = NULL;
    size_t expected_tokens_size = 0;
    read_tuple_u32_u8_array(argv[2],
                            &expected_indices,
                            &expected_indices_size,
                            &expected_tokens,
                            &expected_tokens_size);
    assert(expected_indices_size == expected_tokens_size);
    /*
    size_t input_size = 14;
    uint8_t input[14] = {'t', 'e', 's', 't', ' ', 't', 'e', 's', 't', ' ', 't', 'e', 's', 't'};
    uint32_t expected_indices[5] = {4, 5, 9, 10, 14};
    uint8_t expected_tokens[5] = {1, 0, 1, 0, 1};
    size_t expected_indices_size = 5;
    */
    printf("%s:\n", argv[1]);
    printf(PAD, "Lexer:");
    testLexer(input, input_size, expected_indices, expected_tokens, expected_indices_size);

    free(input);
    free(expected_indices);
    free(expected_tokens);
    gpuAssert(hipPeekAtLastError());
    return 0;
}
