#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "../../common/sps.cu.h"
#include "../../common/util.cu.h"
#include "../../common/data.h"

template<typename I>
struct Add {
    __device__ __forceinline__ I operator()(I a, I b) const {
        return a + b;
    }
};


struct Predicate {
    __device__ inline bool operator()(int32_t a) const {
        return 0 < a;
    }
};

template<typename T>
struct Identity {
    __device__ inline T operator()(T a) const {
        return a;
    }
};

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filter(T* d_in,
       T* d_out,
       volatile State<I>* states,
       I size,
       I num_logical_blocks,
       PRED pred,
       volatile uint32_t* dyn_idx_ptr,
       volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = 0;
        }
    }
    __syncthreads();

    scan<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, states, Add<I>(), 0, dyn_idx);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && ((bools >> i) & 1)) {
            d_out[block[lid] - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = block[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
    }
    __syncthreads();
}

template<typename T, typename I, typename OP, I ITEMS_PER_THREAD>
__device__ inline T
decoupledLookbackScanNoWrite(volatile State<T>* states,
                             volatile T* shmem,
                             OP op,
                             const T ne,
                             uint32_t dyn_idx) {
    volatile __shared__ T values[WARP];
    volatile __shared__ Status statuses[WARP];
    volatile __shared__ T shmem_prefix;
    const uint8_t lane = threadIdx.x & (WARP - 1);
    const bool is_first = threadIdx.x == 0;

    T aggregate = shmem[ITEMS_PER_THREAD * blockDim.x - 1];

    if (is_first) {
        states[dyn_idx].aggregate = aggregate;
    }
    
    if (dyn_idx == 0 && is_first) {
        states[dyn_idx].prefix = aggregate;
    }
    
    __threadfence();
    if (dyn_idx == 0 && is_first) {
        states[dyn_idx].status = Prefix;
    } else if (is_first) {
        states[dyn_idx].status = Aggregate;
    }

    T prefix = ne;
    if (threadIdx.x < WARP && dyn_idx != 0) {
        I lookback_idx = threadIdx.x + dyn_idx;
        I lookback_warp = WARP;
        Status status = Aggregate;
        do {
            if (lookback_warp <= lookback_idx) {
                I idx = lookback_idx - lookback_warp;
                status = states[idx].status;
                statuses[threadIdx.x] = status;
                values[threadIdx.x] = status == Prefix ? states[idx].prefix : states[idx].aggregate;
            } else {
                statuses[threadIdx.x] = Aggregate;
                values[threadIdx.x] = ne;
            }

            scanWarp<T, I, OP>(values, statuses, op, lane);

            T result = values[WARP - 1];
            status = statuses[WARP - 1];

            if (status == Invalid)
                continue;
                
            if (is_first) {
                prefix = op(result, prefix);
            }

            lookback_warp += WARP;
        } while (status != Prefix);
    }

    if (is_first) {
        shmem_prefix = prefix;
    }

    __syncthreads();

    if (is_first) {
        states[dyn_idx].prefix = op(prefix, aggregate);
        __threadfence();
        states[dyn_idx].status = Prefix;
    }
    
    return shmem_prefix;
}

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterFewerShmemWrite(T* d_in,
                      T* d_out,
                      volatile State<I>* states,
                      I size,
                      I num_logical_blocks,
                      PRED pred,
                      volatile uint32_t* dyn_idx_ptr,
                      volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = 0;
        }
    }
    __syncthreads();

    scanBlock<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, Add<I>());

    I prefix = decoupledLookbackScanNoWrite<I, I, Add<I>, ITEMS_PER_THREAD>(states, block, Add<I>(), I(), dyn_idx);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && ((bools >> i) & 1)) {
            d_out[Add<I>()(prefix, block[lid]) - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = Add<I>()(prefix, block[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    }
    __syncthreads();
}

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterCoalescedWrite(T* d_in,
                     T* d_out,
                     volatile State<I>* states,
                     I size,
                     I num_logical_blocks,
                     PRED pred,
                     volatile uint32_t* dyn_idx_ptr,
                     volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    volatile __shared__ I block_keep_size;
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;
    I local_offsets[ITEMS_PER_THREAD];

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = I();
        }
    }
    __syncthreads();

    scanBlock<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, Add<I>());

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        local_offsets[i] = block[lid];
    }

    if (threadIdx.x == blockDim.x - 1) {
        block_keep_size = block[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
    }
    __syncthreads();

    I prefix = decoupledLookbackScanNoWrite<I, I, Add<I>, ITEMS_PER_THREAD>(states, block, Add<I>(), I(), dyn_idx);

    T *block_cast = (T*) &block;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if ((bools >> i) & 1) {
            block_cast[local_offsets[i] - 1] = elems[i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if (lid < block_keep_size) {
            elems[i] = block_cast[lid];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if ((bools >> i) & 1) {
            block[local_offsets[i] - 1] = local_offsets[i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (lid < block_keep_size) {
            d_out[Add<I>()(prefix, block[lid]) - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = Add<I>()(prefix, block[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    }
    __syncthreads();
}

void testFilter(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 10;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    timeval * temp = (timeval *) malloc(sizeof(timeval) * RUNS);
    timeval prev;
    timeval curr;
    timeval t_diff;

    for (I i = 0; i < RUNS; ++i) {
        gettimeofday(&prev, NULL);
        filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        gettimeofday(&curr, NULL);
        timeval_subtract(&t_diff, &curr, &prev);
        temp[i] = t_diff;
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    free(temp);

    filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << "\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Test Failed: Due to elements mismatch at index=" << i << "\n";
            }
        } 
    }

    if (test_passes) {
        std::cout << "Filter test passed." << "\n";
    }

    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

void testFilterCoalescedWrite(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 10;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    timeval * temp = (timeval *) malloc(sizeof(timeval) * RUNS);
    timeval prev;
    timeval curr;
    timeval t_diff;

    for (I i = 0; i < RUNS; ++i) {
        gettimeofday(&prev, NULL);
        filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        gettimeofday(&curr, NULL);
        timeval_subtract(&t_diff, &curr, &prev);
        temp[i] = t_diff;
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    free(temp);

    filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Coalesced Write Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << "\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Coalesced Write Test Failed: Due to elements mismatch at index=" << i << "\n";
            }
        } 
    }

    if (test_passes) {
        std::cout << "Filter Coalesced Write test passed." << "\n";
    }

    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

void testFilterFewerShmemWrite(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 10;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    timeval * temp = (timeval *) malloc(sizeof(timeval) * RUNS);
    timeval prev;
    timeval curr;
    timeval t_diff;

    for (I i = 0; i < RUNS; ++i) {
        gettimeofday(&prev, NULL);
        filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        gettimeofday(&curr, NULL);
        timeval_subtract(&t_diff, &curr, &prev);
        temp[i] = t_diff;
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    free(temp);

    filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter With Fewer Shared Memory Writes Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << "\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter With Fewer Shared Memory Writes Test Failed: Due to elements mismatch at index=" << i << "\n";
            }
        } 
    }

    if (test_passes) {
        std::cout << "Filter With Fewer Shared Memory Writes test passed." << "\n";
    }

    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

int main(int32_t argc, char *argv[]) {
    assert(argc == 3);
    size_t input_size;
    int32_t* input = read_i32_array(argv[1], &input_size);
    size_t expected_size;
    int32_t* expected = read_i32_array(argv[2], &expected_size);
    testFilter(input, input_size, expected, expected_size);
    printf("\n");
    testFilterCoalescedWrite(input, input_size, expected, expected_size);
    printf("\n");
    testFilterFewerShmemWrite(input, input_size, expected, expected_size);
    free(input);
    free(expected);

    std::cout << std::flush;

    gpuAssert(hipPeekAtLastError());
    return 0;
}
