#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "../../common/sps.cu.h"
#include "../../common/util.cu.h"
#include "../../common/data.h"
#include <unistd.h>
#define PAD "%-42s "

template<typename I>
struct Add {
    __device__ __forceinline__ I operator()(I a, I b) const {
        return a + b;
    }
};


struct Predicate {
    __device__ inline bool operator()(int32_t a) const {
        return 0 < a;
    }
};

template<typename T>
struct Identity {
    __device__ inline T operator()(T a) const {
        return a;
    }
};

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filter(T* d_in,
       T* d_out,
       volatile State<I>* states,
       I size,
       I num_logical_blocks,
       PRED pred,
       volatile uint32_t* dyn_idx_ptr,
       volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = 0;
        }
    }
    __syncthreads();

    scan<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, states, Add<I>(), 0, dyn_idx);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && ((bools >> i) & 1)) {
            d_out[block[lid] - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = block[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
    }
    __syncthreads();
}

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterTwoKernel1(T* d_in,
                 I* d_out,
                 I* d_flags_out,
                 volatile State<I>* states,
                 I size,
                 I num_logical_blocks,
                 PRED pred,
                 volatile uint32_t* dyn_idx_ptr) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            T elem = d_in[i];
            I flag = pred(elem);
            block[lid] = flag;
            d_flags_out[gid] = flag;
        } else {
            block[lid] = 0;
        }
    }
    __syncthreads();

    scan<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, states, Add<I>(), 0, dyn_idx);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            d_out[gid] = block[lid];
        }
    }
}

template<typename T, typename I, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterTwoKernel2(T* d_in,
                 I* d_flags,
                 I* d_offset,
                 T* d_out,
                 I size) {
    volatile __shared__ T elem_block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I flags_block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I d_offset_block[ITEMS_PER_THREAD * BLOCK_SIZE];
    I glb_offs = blockIdx.x * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            flags_block[lid] = d_flags[gid];
        } else {
            flags_block[lid] = false;
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elem_block[lid] = d_in[gid];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            d_offset_block[lid] = d_offset[gid];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && flags_block[lid]) {
            d_out[d_offset_block[lid] - 1] = elem_block[lid];
        }
    }
}

template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterFewerShmemWrite(T* d_in,
                      T* d_out,
                      volatile State<I>* states,
                      I size,
                      I num_logical_blocks,
                      PRED pred,
                      volatile uint32_t* dyn_idx_ptr,
                      volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = 0;
        }
    }
    __syncthreads();

    scanBlock<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, Add<I>());

    I prefix = decoupledLookbackScanNoWrite<I, I, Add<I>, ITEMS_PER_THREAD>(states, block, Add<I>(), I(), dyn_idx);

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size && ((bools >> i) & 1)) {
            d_out[Add<I>()(prefix, block[lid]) - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = Add<I>()(prefix, block[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    }
    __syncthreads();
}


template<typename T, typename I, typename PRED, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
filterCoalescedWrite(T* d_in,
                     T* d_out,
                     volatile State<I>* states,
                     I size,
                     I num_logical_blocks,
                     PRED pred,
                     volatile uint32_t* dyn_idx_ptr,
                     volatile I* new_size) {
    volatile __shared__ I block[ITEMS_PER_THREAD * BLOCK_SIZE];
    volatile __shared__ I block_aux[BLOCK_SIZE];
    I block_keep_size;
    T elems[ITEMS_PER_THREAD];
    uint32_t bools = 0;
    I local_offsets[ITEMS_PER_THREAD];

    uint32_t dyn_idx = dynamicIndex<uint32_t>(dyn_idx_ptr);
    I glb_offs = dyn_idx * BLOCK_SIZE * ITEMS_PER_THREAD;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = i * blockDim.x + threadIdx.x;
        I gid = glb_offs + lid;
        if (gid < size) {
            elems[i] = d_in[gid];
            bool temp = pred(elems[i]);
            bools |= temp << i;
            block[lid] = temp;
        } else {
            block[lid] = I();
        }
    }
    __syncthreads();

    scanBlock<I, I, Add<I>, ITEMS_PER_THREAD>(block, block_aux, Add<I>());

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        local_offsets[i] = block[lid];
    }

    block_keep_size = block[ITEMS_PER_THREAD * BLOCK_SIZE - 1];
    
    __syncthreads();

    I prefix = decoupledLookbackScanNoWrite<I, I, Add<I>, ITEMS_PER_THREAD>(states, block, Add<I>(), I(), dyn_idx);

    T *block_cast = (T*) &block;

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if ((bools >> i) & 1) {
            block_cast[local_offsets[i] - 1] = elems[i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if (lid < block_keep_size) {
            elems[i] = block_cast[lid];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        if ((bools >> i) & 1) {
            block[local_offsets[i] - 1] = local_offsets[i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (I i = 0; i < ITEMS_PER_THREAD; i++) {
        I lid = blockDim.x * i + threadIdx.x;
        I gid = glb_offs + lid;
        if (lid < block_keep_size) {
            d_out[Add<I>()(prefix, block[lid]) - 1] = elems[i];
        }
    }
    
    if (dyn_idx == num_logical_blocks - 1 && threadIdx.x == blockDim.x - 1) {
        *new_size = Add<I>()(prefix, block[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    }
    __syncthreads();
}

void testFilter(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 500;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    float * temp = (float *) malloc(sizeof(float) * RUNS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    for (I i = 0; i < RUNS; ++i) {
        hipEventRecord(start, 0);
        filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(temp + i, start, stop);
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    filter<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << ".\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Test Failed: Due to elements mismatch at index=" << i << ".\n";
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    }

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

void testFilterCoalescedWrite(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 512;
    const I ITEMS_PER_THREAD = 14;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 500;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    float * temp = (float *) malloc(sizeof(float) * RUNS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    for (I i = 0; i < RUNS; ++i) {
        hipEventRecord(start, 0);
        filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(temp + i, start, stop);
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;

    filterCoalescedWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Coalesced Write Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << ".\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Coalesced Write Test Failed: Due to elements mismatch at index=" << i << ".\n";
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    }

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

void testFilterFewerShmemWrite(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 30;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 500;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_new_size;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    float * temp = (float *) malloc(sizeof(float) * RUNS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    for (I i = 0; i < RUNS; ++i) {
        hipEventRecord(start, 0);
        filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(temp + i, start, stop);
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    filterFewerShmemWrite<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(d_in, d_out, d_states, size, NUM_LOGICAL_BLOCKS, pred, d_dyn_idx_ptr, d_new_size);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter With Fewer Shared Memory Writes Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << ".\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter With Fewer Shared Memory Writes Test Failed: Due to elements mismatch at index=" << i << ".\n";
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    }

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
    gpuAssert(hipFree(d_new_size));
}

void testFilterCUB(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 500;
    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    I* d_new_size;
    int32_t *d_in, *d_out;
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_new_size, size, pred);

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    float * temp = (float *) malloc(sizeof(float) * RUNS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_new_size, size, pred);
        hipDeviceSynchronize();
        gpuAssert(hipPeekAtLastError());
    }

    for (I i = 0; i < RUNS; ++i) {
        hipEventRecord(start, 0);
        hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_new_size, size, pred);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(temp + i, start, stop);
        hipMemset(d_new_size, 0, sizeof(I));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;

    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_new_size, size, pred);
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(&temp_size, d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter (CUB) Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << ".\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter (CUB) Test Failed: Due to elements mismatch at index=" << i << ".\n";
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    }

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_new_size));
    gpuAssert(hipFree(d_temp_storage));
}

void testFilterTwoKernels(int32_t* input, size_t input_size, int32_t* expected, size_t expected_size) {
    using I = uint32_t;
    const I size = input_size;
    const I BLOCK_SIZE = 256;
    const I ITEMS_PER_THREAD = 4;
    const I NUM_LOGICAL_BLOCKS = (size + BLOCK_SIZE * ITEMS_PER_THREAD - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);
    const I ARRAY_BYTES = size * sizeof(int32_t);
    const I OFFSETS_BYTES = size * sizeof(I);
    const I FLAGS_BYTES = size * sizeof(I);
    const I STATES_BYTES = NUM_LOGICAL_BLOCKS * sizeof(State<I>);
    const I WARMUP_RUNS = 1000;
    const I RUNS = 500;
    assert(ITEMS_PER_THREAD <= 32);

    std::vector<int32_t> h_in(size);
    std::vector<int32_t> h_out(size, 0);
    std::vector<I> h_offsets(size, 0);

    for (I i = 0; i < size; ++i) {
        h_in[i] = input[i];
    }
    
    uint32_t* d_dyn_idx_ptr;
    I* d_flags;
    I* d_offsets;
    int32_t *d_in, *d_out;
    State<I>* d_states;
    gpuAssert(hipMalloc((void**)&d_dyn_idx_ptr, sizeof(uint32_t)));
    hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
    gpuAssert(hipMalloc((void**)&d_states, STATES_BYTES));
    gpuAssert(hipMalloc((void**)&d_in, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_out, ARRAY_BYTES));
    gpuAssert(hipMalloc((void**)&d_flags, FLAGS_BYTES));
    gpuAssert(hipMalloc((void**)&d_offsets, OFFSETS_BYTES));
    
    gpuAssert(hipMemcpy(d_in, h_in.data(), ARRAY_BYTES, hipMemcpyHostToDevice));
    
    Predicate pred = Predicate();
    
    float * temp = (float *) malloc(sizeof(float) * RUNS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (I i = 0; i < WARMUP_RUNS; ++i) {
        filterTwoKernel1<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            d_in,
            d_offsets,
            d_flags,
            d_states,
            size,
            NUM_LOGICAL_BLOCKS,
            pred,
            d_dyn_idx_ptr
        );
        hipDeviceSynchronize();
        filterTwoKernel2<int32_t, I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            d_in,
            d_flags,
            d_offsets,
            d_out,
            size
        );
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    for (I i = 0; i < RUNS; ++i) {
        hipEventRecord(start, 0);
        filterTwoKernel1<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            d_in,
            d_offsets,
            d_flags,
            d_states,
            size,
            NUM_LOGICAL_BLOCKS,
            pred,
            d_dyn_idx_ptr
        );
        hipDeviceSynchronize();
        filterTwoKernel2<int32_t, I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
            d_in,
            d_flags,
            d_offsets,
            d_out,
            size
        );
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(temp + i, start, stop);
        hipMemset(d_dyn_idx_ptr, 0, sizeof(uint32_t));
        gpuAssert(hipPeekAtLastError());
    }

    I temp_size = 0;
    filterTwoKernel1<int32_t, I, Predicate, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
        d_in,
        d_offsets,
        d_flags,
        d_states,
        size,
        NUM_LOGICAL_BLOCKS,
        pred,
        d_dyn_idx_ptr
    );
    hipDeviceSynchronize();
    filterTwoKernel2<int32_t, I, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_LOGICAL_BLOCKS, BLOCK_SIZE>>>(
        d_in,
        d_flags,
        d_offsets,
        d_out,
        size
    );
    hipDeviceSynchronize();
    gpuAssert(hipMemcpy(h_out.data(), d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy(h_offsets.data(), d_offsets, OFFSETS_BYTES, hipMemcpyDeviceToHost));
    temp_size = h_offsets[size - 1];
    
    bool test_passes = temp_size == expected_size;
    if (!test_passes) {
        std::cout << "Filter Test Failed: Expected size=" << expected_size << " but got size=" << temp_size << ".\n";
    } else {
        for (I i = 0; i < expected_size; ++i) {
            test_passes &= h_out[i] == expected[i];

            if (!test_passes) {
                std::cout << "Filter Test Failed: Due to elements mismatch at index=" << i << ".\n";
            }
        } 
    }

    if (test_passes) {
        compute_descriptors(temp, RUNS, ARRAY_BYTES + temp_size * sizeof(int32_t));
    }

    free(temp);
    gpuAssert(hipFree(d_in));
    gpuAssert(hipFree(d_flags));
    gpuAssert(hipFree(d_offsets));
    gpuAssert(hipFree(d_out));
    gpuAssert(hipFree(d_states));
    gpuAssert(hipFree(d_dyn_idx_ptr));
}


int main(int32_t argc, char *argv[]) {
    assert(argc == 3);
    size_t input_size;
    int32_t* input = read_i32_array(argv[1], &input_size);
    size_t expected_size;
    int32_t* expected = read_i32_array(argv[2], &expected_size);
    printf("%s:\n", argv[1]);
    /*
    printf(PAD, "Filter:");
    testFilter(input, input_size, expected, expected_size);
    printf(PAD, "Filter Coalesced Write:");
    testFilterCoalescedWrite(input, input_size, expected, expected_size);
    printf(PAD, "Filter With Fewer Shared Memory Writes:");
    testFilterFewerShmemWrite(input, input_size, expected, expected_size);
    printf(PAD, "Filter (CUB):");
    testFilterCUB(input, input_size, expected, expected_size);
    */
    printf(PAD, "Filter Two Kernels:");
    testFilterTwoKernels(input, input_size, expected, expected_size);
    free(input);
    free(expected);

    gpuAssert(hipPeekAtLastError());
    return 0;
}
